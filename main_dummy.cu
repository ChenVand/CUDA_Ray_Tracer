#include "hip/hip_runtime.h"
/*
cmake -B build
cmake --build build
build/inOneWeekend > image.ppm
*/ 

// cspell: disable

#include <stdio.h>
// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


#include "rtweekend.h"

#include "hittable.h"
#include "hittable_list.h"
#include "sphere.h"

__device__ color ray_color(const ray& r, const sphere test_sphere) {

    //debug
    printf("reached ray_color before hit check\n");

    hit_record* rec = new hit_record;
    if (test_sphere.hit(r, 0, infinity, rec)) {
        return 0.5 * (rec->normal + color(1,1,1));
    }
    
    //debug
    printf("reached ray_color after hit check\n");

    vec3 unit_direction = unit_vector(r.direction());
    float a = 0.5f*(unit_direction.y() + 1.0f);
    return (1.0f-a)*color(1.0, 1.0, 1.0) + a*color(0.5, 0.7, 1.0);
}

__global__ void render_test_sphere(vec3 *fb, int max_x, int max_y, const vec3 *cam_deets, const sphere* test_sphere) {

    __shared__ sphere* local_sphere; // Declare shared memory
    *local_sphere = *test_sphere;
        
    /*cam_deets: pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center*/
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if((x >= max_x) || (y >= max_y)) return;
    int pixel_index = y*max_x + x;

    auto pixel_center = cam_deets[0] + (x * cam_deets[1]) + (y * cam_deets[2]);
    auto ray_direction = pixel_center - cam_deets[3];
    ray r(cam_deets[3], ray_direction);

    color pixel_color = ray_color(r, *local_sphere);

    //debug
    // if (x%10==0 || y%10==0)
    printf("reached renderK for thread %d, %d\n pixel color %f,%f,%f\n", x, y, pixel_color[0], pixel_color[1], pixel_color[2]);

    fb[pixel_index] = pixel_color;

}

__global__ void dummy_kernel() {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    //debug
    if (x%10==0 && y%10==0)
    printf("reached here in dummy kernel for thread %d, %d\n", x, y);
}

int main(int argc,char *argv[]) {

    hipError_t err = hipSuccess;

    // Image
    int image_width = (argc >1) ? atoi(argv[1]) : 16;
    auto aspect_ratio = 16.0 / 9.0;

    // Calculate the image height, and ensure that it's at least 1.
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;

    // World
    hittable_list* world;   
    hipMallocManaged(&world, sizeof(hittable_list)); 
    cudaCheckErrors("world managed mem alloc failure");
    new (world) hittable_list(); // Placement new to call the constructor
    cudaCheckErrors("initialization error");

    int num_spheres = 2;
    sphere* spheres;
    hipMallocManaged(&spheres, num_spheres*sizeof(hittable_list));
    cudaCheckErrors("spheres managed mem alloc failure");
    // spheres[0] = sphere(point3(0,0,-1), 0.5);
    // spheres[1] = sphere(point3(0,-100.5,-1), 100);
    new (&spheres[0]) sphere(point3(0, 0, -1), 0.5); // Placement new to call the constructor
    new (&spheres[1]) sphere(point3(0, -100.5, -1), 100); // Placement new to call the constructor
    cudaCheckErrors("initialization error");

    for (int i = 0; i < num_spheres; i++) {
        world->add(&spheres[i]);
    }
    cudaCheckErrors("initialization error");

    //Test sphere
    sphere* test_sphere;
    hipMallocManaged(&test_sphere, sizeof(hittable_list));
    cudaCheckErrors("spheres managed mem alloc failure");
    new (test_sphere) sphere(point3(0, 0, -1), 0.5); // Placement new to call the constructor
    cudaCheckErrors("initialization error");

    // Camera

    auto focal_length = 1.0;
    auto viewport_height = 2.0;
    auto viewport_width = viewport_height * (double(image_width)/image_height);
    auto camera_center = point3(0, 0, 0);

    // Calculate the vectors across the horizontal and down the vertical viewport edges.
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);

    // Calculate the horizontal and vertical delta vectors from pixel to pixel.
    auto pixel_delta_u = viewport_u / image_width;
    auto pixel_delta_v = viewport_v / image_height;

    // Calculate the location of the upper left pixel.
    auto viewport_upper_left = camera_center
                             - vec3(0, 0, focal_length) - viewport_u/2 - viewport_v/2;
    auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);

    // Render

    int num_pixels = image_width*image_height;

    //cam_deets: pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center
    vec3* cam_deets;
    hipMallocManaged(&cam_deets, 4*sizeof(vec3));
    cudaCheckErrors("cam_deets managed mem alloc failure");
    cam_deets[0] = pixel00_loc;
    cam_deets[1] = pixel_delta_u;
    cam_deets[2] = pixel_delta_v;
    cam_deets[3] = camera_center;

    // allocate frame buffer
    size_t fb_size = num_pixels*sizeof(vec3);
    vec3 *fb;
    // hipMalloc(&fb, fb_size);
    // hipMemcpy(d_cam_deets, &h_cam_deets, 4 * sizeof(vec3), hipMemcpyHostToDevice);
    hipMallocManaged(&fb, fb_size);
    cudaCheckErrors("frame buffer managed mem alloc failure");

    // block size
    int tx = 32;
    int ty = 8;

    // Render our buffer
    // dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 blocks(1,1);
    dim3 threads(tx,ty);
    hipMemPrefetchAsync(fb, fb_size, 0);
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Device synchronization 0 failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    render_test_sphere<<<blocks, threads>>>(fb, image_width, image_height, cam_deets, test_sphere);
    // cudaCheckErrors("render kernel launch failure");
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Device synchronization failed: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    hipMemPrefetchAsync(fb, fb_size, hipCpuDeviceId);

    // Cleanup
    // world->clear();
    hipFree(fb);
    hipFree(spheres);
    // hipFree(cam_deets);
    
    return 0;
}