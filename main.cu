#include "hip/hip_runtime.h"
// cspell: disable

#include <stdio.h>
#include <time.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
// #include <thrust/host_vector.h>
// #include <thrust/device_vector.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

#include "rtweekend.h"

#include "hittable.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"

__global__ void generate_randoms(hiprandState_t* state, float* randoms) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState localState = state[tid];
    randoms[tid] = hiprand_uniform(&localState);
}



__global__ void create_world(hittable** world, hittable** objects, int num_objects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        objects[0] = new sphere(point3(0, 0, -1), 0.5);
        objects[1] = new sphere(point3(0, -100.5, -1), 100);
        *world = new hittable_list(objects, num_objects);
    }
}

__global__ void destroy_world(hittable** world, hittable** objects, int num_objects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *world;
        for (int i = 0; i < num_objects; i++) {
            delete objects[i];
        }
    }
}

int main(int argc,char *argv[]) {
    /*exe_name image_width threads_per_block_x threads_per_block_y*/

    // Camera preparation

    camera cam;

    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_width  = (argc >1) ? atoi(argv[1]) : 400;
    cam.samples_per_pixel = (argc >2) ? atoi(argv[2]) : 32; //streches block x dim
    cam.initialize();

    // World

    // device memory allocation for world and objects
    int num_objects = 2;
    hittable** world;
    hipMalloc((void **)&world, sizeof(hittable*));
    hittable** objects;
    hipMalloc((void **)&objects, sizeof(hittable*) * num_objects);
    create_world<<<1,1>>>(world, objects, num_objects);
    hipDeviceSynchronize();
    cudaCheckErrors("post-world-creation synchronization failed");

    // Render

    int pixels_per_block_x = (argc >3) ? atoi(argv[3]) : 4; //blockDim.x will be this times samples_per_pixel
    int pixels_per_block_y = (argc >4) ? atoi(argv[4]) : 8;
    float buffer_gen_time;

    std::cerr << "Rendering width " << cam.image_width << " image ";
    std::cerr << "with " << pixels_per_block_x*cam.samples_per_pixel << 
        "x" << pixels_per_block_y << " blocks.\n";

    cam.render(pixels_per_block_x, pixels_per_block_y, world, buffer_gen_time);
    
    std::cerr << "Buffer creation took " << buffer_gen_time << " seconds.\n";

    // Cleanup

    hipDeviceSynchronize();
    cudaCheckErrors("final synchronization failed");
    destroy_world<<<1,1>>>(world,
        objects,
        num_objects);
    hipFree(world);
    hipFree(objects);

}