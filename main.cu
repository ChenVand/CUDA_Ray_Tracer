#include "hip/hip_runtime.h"
// cspell: disable

#include <stdio.h>
#include <time.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


#include "rtweekend.h"
#include "hittable.h"
#include "material.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"

__global__ void generate_randoms(hiprandState_t* state, float* randoms) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState localState = state[tid];
    randoms[tid] = hiprand_uniform(&localState);
}

__global__ void create_world(hittable** world, material_list** mat_lst) {    //}, hittable** objects, int num_objects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {

        // Materials
        const int num_materials = 5;
        material** materials = new material*[num_materials];

        materials[0] = new lambertian(color(0.8, 0.2, 0.2)); //ground
        materials[1] = new lambertian(color(0.1, 0.2, 0.5)); //center
        materials[2] = new dielectric(1.50); //left
        materials[3] = new dielectric(1.00 / 1.50); //bubble
        materials[4] = new metal(color(0.7, 0.7, 0.7), 0.2); //right

        *mat_lst = new material_list(materials, num_materials); //"Owner" list


        // Objects
        const int num_objects = 5;
        hittable** objects = new hittable*[num_objects];

        objects[0] = new sphere(point3( 0.0, -100.5, -1.0), 100.0, materials[0]); //ground
        objects[1] = new sphere(point3( 0.0,    0.0, -1.2),   0.5, materials[1]); //center
        objects[2] = new sphere(point3( -1.0,   0.0, -1.0),   0.5, materials[2]); //left
        objects[3] = new sphere(point3( -1.0,   0.0, -1.0),   0.4, materials[3]); //bubble
        objects[4] = new sphere(point3( 1.0,    0.0, -1.0),   0.5, materials[4]); //right

        *world = new hittable_list(objects, num_objects);
    }
}

__global__ void create_world2(hittable** world, material_list** mat_lst) {    //}, hittable** objects, int num_objects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {

        float R = cosf(pi/4);

        // Materials
        const int num_materials = 2;
        material** materials = new material*[num_materials];

        materials[0] = new lambertian(color(0,0,1));
        materials[1] = new lambertian(color(1,0,0));

        *mat_lst = new material_list(materials, num_materials); //"Owner" list


        // Objects
        const int num_objects = 2;
        hittable** objects = new hittable*[num_objects];

        objects[0] = new sphere(point3(-R, 0, -1), R, materials[0]); //ground
        objects[1] = new sphere(point3( R, 0, -1), R, materials[1]); //center
        
        *world = new hittable_list(objects, num_objects);
    }
}

__global__ void destroy_world(hittable** world, material_list** mat_lst) {   //}, hittable** objects, int num_objects) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        delete *world;
        delete *mat_lst;
    }
}

// Tunable variables

// extern bool g_lambertian = true; //Try again by making constant
size_t g_image_width = 400;
size_t g_samples_per_pixel = 32;
int g_threads_x = 2 * g_samples_per_pixel;
int g_threads_y = 8;

int main(int argc,char *argv[]) {
    /*exe_name image_width threads_per_block_x threads_per_block_y*/

    // External arguments
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], "--width") == 0 && i + 1 < argc) {
            g_image_width = atoi(argv[i + 1]);
            i++; // Skip the next argument as it is the value
        } else if (strcmp(argv[i], "--samples") == 0 && i + 1 < argc) {
            g_samples_per_pixel = atoi(argv[i + 1]);
            i++; // Skip the next argument as it is the value
        } else if (strcmp(argv[i], "--threads") == 0 && i + 2 < argc) {
            g_threads_x = atoi(argv[i + 1]);
            g_threads_y = atoi(argv[i + 2]);
            i+=2; // Skip the next argument as it is the value
        } 
        // else if (strcmp(argv[i], "--lambertian") == 0 && i + 1 < argc) {
        //     g_lambertian = !(strcmp(argv[i + 1], "false") == 0);
        //     i++; // Skip the next argument as it is the value
        // } 
        else {
            std::cerr << "Unknown argument: " << argv[i] << "\n";
            return 1;
        }
    }

    // Camera preparation

    camera cam;

    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_width  = g_image_width;
    cam.samples_per_pixel = g_samples_per_pixel; //streches block x dim
    //cam.max_depth = 50; // Not used in this version

    cam.vfov = 20; // Zoom with range >0 (close up) to <180 (far away)
    cam.lookfrom = point3(-2,2,1);
    cam.lookat   = point3(0,0,-1);
    cam.vup      = vec3(0,1,0);

    cam.defocus_angle = 10.0;
    cam.focus_dist    = 3.4;

    cam.initialize();

    // World

    hittable** world;
    hipMalloc((void **)&world, sizeof(hittable*));
    material_list** mat_lst; //material packet for deallocation
    hipMalloc((void **)&mat_lst, sizeof(material_list*));

    create_world<<<1,1>>>(world, mat_lst);
    cudaCheckErrors("create world kernel launch failed");
    hipDeviceSynchronize();
    cudaCheckErrors("post-world-creation synchronization failed");

    // Render

    int pixels_per_block_x = (g_threads_x + g_samples_per_pixel - 1)/g_samples_per_pixel; //blockDim.x will be this times samples_per_pixel
    int pixels_per_block_y = g_threads_y;

    std::cerr << "Rendering width " << cam.image_width << " image ";
    std::cerr << "with " << pixels_per_block_x*cam.samples_per_pixel << 
        "x" << pixels_per_block_y << " blocks.\n";

    float buffer_gen_time;
    cam.render(pixels_per_block_x, pixels_per_block_y, world, buffer_gen_time);
    
    std::cerr << "Buffer creation took " << buffer_gen_time << " seconds.\n";

    // Cleanup

    hipDeviceSynchronize();
    cudaCheckErrors("final synchronization failed");
    destroy_world<<<1,1>>>(world, mat_lst);
    cudaCheckErrors("destroy world kernel launch failed");
    hipFree(world);
    hipFree(mat_lst);

}