/*
cmake -B build
cmake --build build
build/inOneWeekend > image.ppm
*/ 
////

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

// #include "thrust/host_vector.h"
// #include "thrust/device_vector.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}

int main() {

    // Image

    int image_width = 256;
    int image_height = 256;

    // Render
    int nx = image_width;
    int ny = image_height;

    int num_pixels = nx*ny;
    size_t fb_size = 3*num_pixels*sizeof(float);

    // allocate frame buffer
    float *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // block size
    int tx = 8;
    int ty = 8;

    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Print

    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    for (int j = 0; j < image_height; j++) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j*3*nx + i*3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];

            int ir = int(255.999 * r);
            int ig = int(255.999 * g);
            int ib = int(255.999 * b);

            std::cout << ir << ' ' << ig << ' ' << ib << '\n';
        }
    }
    checkCudaErrors(hipFree(fb));
}