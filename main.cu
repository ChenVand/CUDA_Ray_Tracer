#include "hip/hip_runtime.h"
/*
cmake -B build
cmake --build build
build/inOneWeekend > image.ppm
*/ 

#include <stdio.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "color.h"
#include "vec3.h"
#include "ray.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ color ray_color(const ray& r) {
    vec3 unit_direction = unit_vector(r.direction());
    float a = 0.5f*(unit_direction.y() + 1.0f);
    return (1.0f-a)*color(1.0, 1.0, 1.0) + a*color(0.5, 0.7, 1.0);
}

__global__ void render(vec3 *fb, int max_x, int max_y, const vec3 *cam_deets) {
    /*cam_deets: pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center*/
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;

    auto pixel_center = cam_deets[0] + (i * cam_deets[1]) + (j * cam_deets[2]);
    auto ray_direction = pixel_center - cam_deets[3];
    ray r(cam_deets[3], ray_direction);

    color pixel_color = ray_color(r);
    fb[pixel_index] = pixel_color;
}

int main() {

    // Image

    auto aspect_ratio = 16.0 / 9.0;
    int image_width = 400;

    // Calculate the image height, and ensure that it's at least 1.
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;

    // Camera

    auto focal_length = 1.0;
    auto viewport_height = 2.0;
    auto viewport_width = viewport_height * (double(image_width)/image_height);
    auto camera_center = point3(0, 0, 0);

    // Calculate the vectors across the horizontal and down the vertical viewport edges.
    auto viewport_u = vec3(viewport_width, 0, 0);
    auto viewport_v = vec3(0, -viewport_height, 0);

    // Calculate the horizontal and vertical delta vectors from pixel to pixel.
    auto pixel_delta_u = viewport_u / image_width;
    auto pixel_delta_v = viewport_v / image_height;

    // Calculate the location of the upper left pixel.
    auto viewport_upper_left = camera_center
                             - vec3(0, 0, focal_length) - viewport_u/2 - viewport_v/2;
    auto pixel00_loc = viewport_upper_left + 0.5 * (pixel_delta_u + pixel_delta_v);

    // Render

    int num_pixels = image_width*image_height;

    //cam_deets: pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center
    vec3 h_cam_deets[4] = {pixel00_loc, pixel_delta_u, pixel_delta_v, camera_center};
    vec3* d_cam_deets;
    hipMalloc(&d_cam_deets, 4 * sizeof(vec3));
    hipMemcpy(d_cam_deets, &h_cam_deets, 4 * sizeof(vec3), hipMemcpyHostToDevice);

    // allocate frame buffer
    size_t fb_size = num_pixels*sizeof(vec3);
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // block size
    int tx = 8;
    int ty = 8;

    // Render our buffer
    dim3 blocks(image_width/tx+1,image_height/ty+1);
    dim3 threads(tx,ty);
    render<<<blocks, threads>>>(fb, image_width, image_height, d_cam_deets);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Print

    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";

    for (int j = 0; j < image_height; j++) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j*image_width + i;
            auto pixel_color = fb[pixel_index];

            write_color(std::cout, pixel_color);
        }
    }
    checkCudaErrors(hipFree(fb));
    checkCudaErrors(hipFree(d_cam_deets));
}